#include "hip/hip_runtime.h"
#include <iostream>
#include "utility/utils.cuh"
#include "kernels/partitioning_kernels.cuh"
#include "include/partitioning_impl.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <sys/time.h>


void partition_opt2(Hypergraph* hgr, unsigned int K, bool use_curr_precision, float& time) {
    std::cout << __FUNCTION__ << "()===========\n";
    TIMERSTART(_)
    thrust::fill(thrust::device, hgr->nodes + N_PARTITION(hgr), hgr->nodes + N_PARTITION(hgr) + hgr->nodeNum, 1);
    TIMERSTOP(_)
    struct timeval bbeg, eend;
    int zeroPartCnt = 0;
    // std::ofstream out0("zeroPart.txt");
    // std::ofstream out1("nonzeroPart.txt");
    gettimeofday(&bbeg, NULL);
    for (int i = 0; i < hgr->hedgeNum; ++i) {
        for (int j = 0; j < hgr->hedges[i + E_DEGREE(hgr)]; ++j) {
            hgr->nodes[hgr->adj_list[hgr->hedges[i + E_OFFSET(hgr)] + j]-hgr->hedgeNum + N_PARTITION(hgr)] = 0;
        }
    }
    int zeroPartWeight = 0;
    for (int i = 0; i < hgr->nodeNum; ++i) {
        if (hgr->nodes[i + N_PARTITION(hgr)] == 0) {
            if (hgr->nodes[i + N_WEIGHT(hgr)] == hgr->maxWeight) {
                continue;
            }
            zeroPartWeight += hgr->nodes[i + N_WEIGHT(hgr)];
            zeroPartCnt++;
            // out0 << i << ": " << hgr->nodes[i + N_DEGREE(hgr)] << ", " << hgr->nodes[i + N_WEIGHT(hgr)] << "\n";
            // out0 << i << ": " << hgr->nodes[i + N_WEIGHT(hgr)] << "\n";
        }
        // else {
        //     out1 << i << ": " << hgr->nodes[i + N_DEGREE(hgr)] << ", " << hgr->nodes[i + N_WEIGHT(hgr)] << "\n";
        // }
    }
    gettimeofday(&eend, NULL);
    float elap = (eend.tv_sec - bbeg.tv_sec) + ((eend.tv_usec - bbeg.tv_usec)/1000000.0);
    std::cout << "elapsed time: " << elap << " s.\n";
    time += time_ / 1000.f + elap;

    int onePartWeight = 0;
    for (int i = 0; i < hgr->nodeNum; ++i) {
        if (hgr->nodes[i + N_PARTITION(hgr)] == 1) {
            onePartWeight += hgr->nodes[i + N_WEIGHT(hgr)];
        }
    }
    std::cout << "part1's weight: " << onePartWeight << "\n";
    std::cout << "initially, edge cut quality:" << computeHyperedgeCut(hgr) << "\n";
    std::ofstream debug0("../debug/initial_nodes_partition0.txt");
    std::ofstream debug1("../debug/initial_nodes_partition1.txt");
    for (int i = 0; i < hgr->nodeNum; ++i) {
        if (hgr->nodes[N_PARTITION(hgr) + i] == 0) {
            debug0 << "node " << i << ", weight: " << hgr->nodes[N_WEIGHT(hgr) + i] << ", part: " << hgr->nodes[N_PARTITION(hgr) + i] << "\n";
        } else {
            debug1 << "node " << i << ", weight: " << hgr->nodes[N_WEIGHT(hgr) + i] << ", part: " << hgr->nodes[N_PARTITION(hgr) + i] << "\n";
        }
    }
    int total = hgr->totalWeight - hgr->maxWeight;
    int nonzeroPartWeight = total - zeroPartWeight;
    unsigned kvalue        = (K + 1) / 2;
    unsigned targetWeight0 = total * kvalue / K;
    unsigned targetWeight1 = total - targetWeight0;
    std::cout << "totalweight:" << hgr->totalWeight << ", part0_weight:" << zeroPartWeight << ", part1_weight:" << nonzeroPartWeight << "\n";
    std::cout << "targetWeight0:" << targetWeight0 << ", targetWeight1:" << targetWeight1 << "\n";
    std::cout << "zeroPartCnt:" << zeroPartCnt << "\n";
    int ccount = 0;
    if (static_cast<long>(zeroPartWeight) > nonzeroPartWeight) {
        std::cout << "enter move 0 to 1 branch\n";
        int gain = nonzeroPartWeight;
        std::cout << "initial gain: " << gain << "\n";
        unsigned* count;
        CHECK_ERROR(hipMallocManaged(&count, sizeof(unsigned)));
        while (1) {
            int blocksize = 128;
            int gridsize = UP_DIV(hgr->hedgeNum, blocksize);
            TIMERSTART(0)
            thrust::fill(thrust::device, hgr->nodes + N_FS(hgr), hgr->nodes + N_FS(hgr) + hgr->nodeNum, 0);
            thrust::fill(thrust::device, hgr->nodes + N_TE(hgr), hgr->nodes + N_TE(hgr) + hgr->nodeNum, 0);
            initGain<<<gridsize, blocksize>>>(hgr, hgr->hedgeNum);
            TIMERSTOP(0)

            count[0] = 0;
            tmpNode* nodeListz;
            CHECK_ERROR(hipMallocManaged(&nodeListz, hgr->nodeNum * sizeof(tmpNode)));
            CHECK_ERROR(hipMemset(nodeListz, 0, hgr->nodeNum * sizeof(tmpNode)));

            gridsize = UP_DIV(hgr->nodeNum, blocksize);
            unsigned partID = 0;
            TIMERSTART(1)
            createNodeList<<<gridsize, blocksize>>>(hgr, nodeListz, count, partID);
            TIMERSTOP(1)
            std::cout << count[0] << "\n";

            // thrust::device_ptr<tmpNode> zero_ptr(nodeListz);
            TIMERSTART(2)
            use_curr_precision ? thrust::sort(thrust::device, nodeListz, nodeListz + count[0], mycmp()) : thrust::sort(thrust::device, nodeListz, nodeListz + count[0], mycmp1());
            TIMERSTOP(2)
#if 1
            if (ccount == 0) {
                std::ofstream debug("../debug/initial_partition_nodelist_info.txt");
                for (int i = 0; i < count[0]; ++i) {
                    debug << "nodeid: " << nodeListz[i].nodeid << ", init_gain: " << nodeListz[i].gain << ", real_gain: "
                          << nodeListz[i].real_gain << ", weight: " << nodeListz[i].weight << ", ratio: "
                          << (float)(nodeListz[i].real_gain * (1.0f / nodeListz[i].weight)) << ", degree: " << hgr->nodes[nodeListz[i].nodeid-hgr->hedgeNum + N_DEGREE(hgr)] << "\n";
                }
            }
#endif
            unsigned i = 0;
            struct timeval begin, end;
            gettimeofday(&begin, NULL);
            for (; i < count[0]; ) {
                if (nodeListz[i].weight == hgr->maxWeight) {
                    i++;
                    continue;
                }
                hgr->nodes[nodeListz[i].nodeid-hgr->hedgeNum + N_PARTITION(hgr)] = 1;
                gain += hgr->nodes[nodeListz[i].nodeid-hgr->hedgeNum + N_WEIGHT(hgr)];
                i++;
                if (gain >= static_cast<long>(targetWeight1)) {
                    break;
                }
                if (i > sqrt(hgr->totalWeight)) {
                    break;
                }
            }
            gettimeofday(&end, NULL);
            float elapsed = (end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0);
            time += (time0 + time1 + time2) / 1000.f + elapsed;
            std::cout << "i:" << i << "\n";
            std::cout << "gain:" << gain << ", targetWeight1:" << targetWeight1 << "\n";
            std::cout << "current edge cut quality:" << computeHyperedgeCut(hgr) << "\n";
            CHECK_ERROR(hipFree(nodeListz));
            ccount++;
            if (gain >= static_cast<long>(targetWeight1)) {
                break;
            }
        }
        CHECK_ERROR(hipFree(count));
    } else { // zero < nonzero
        std::cout << "enter move 1 to 0 branch\n";
        int gain = zeroPartWeight;
        unsigned* count;
        CHECK_ERROR(hipMallocManaged(&count, sizeof(unsigned)));
        while (1) {
            int blocksize = 128;
            int gridsize = UP_DIV(hgr->hedgeNum, blocksize);
            TIMERSTART(0)
            thrust::fill(thrust::device, hgr->nodes + N_FS(hgr), hgr->nodes + N_FS(hgr) + hgr->nodeNum, 0);
            thrust::fill(thrust::device, hgr->nodes + N_TE(hgr), hgr->nodes + N_TE(hgr) + hgr->nodeNum, 0);
            initGain<<<gridsize, blocksize>>>(hgr, hgr->hedgeNum);
            TIMERSTOP(0)

            count[0] = 0;
            tmpNode* nodeListnz;
            CHECK_ERROR(hipMallocManaged(&nodeListnz, hgr->nodeNum * sizeof(tmpNode)));
            CHECK_ERROR(hipMemset(nodeListnz, 0, hgr->nodeNum * sizeof(tmpNode)));
            gridsize = UP_DIV(hgr->nodeNum, blocksize);
            unsigned partID = 1;
            TIMERSTART(1)
            createNodeList<<<gridsize, blocksize>>>(hgr, nodeListnz, count, partID);
            TIMERSTOP(1)
            
            std::cout << count[0] << "\n";
            // thrust::device_ptr<tmpNode> one_ptr(nodeListnz);
            TIMERSTART(2)
            use_curr_precision ? thrust::sort(thrust::device, nodeListnz, nodeListnz + count[0], mycmp()) : thrust::sort(thrust::device, nodeListnz, nodeListnz + count[0], mycmp1());
            TIMERSTOP(2)

#if 1
            if (ccount == 0) {
                std::ofstream debug("../debug/initial_partition_nodelist_info.txt");
                for (int i = 0; i < count[0]; ++i) {
                    debug << "nodeid: " << nodeListnz[i].nodeid << ", init_gain: " << nodeListnz[i].gain << ", real_gain: "
                          << nodeListnz[i].real_gain << ", weight: " << nodeListnz[i].weight << ", ratio: "
                          << (float)(nodeListnz[i].real_gain * (1.0f / nodeListnz[i].weight)) << ", degree: " << hgr->nodes[nodeListnz[i].nodeid-hgr->hedgeNum + N_DEGREE(hgr)] << "\n";
                }
            }
#endif

            unsigned i = 0;
            struct timeval begin, end;
            gettimeofday(&begin, NULL);
            for (; i < count[0]; ) {
                if (nodeListnz[i].weight == hgr->maxWeight) {
                    i++;
                    continue;
                }
                hgr->nodes[nodeListnz[i].nodeid-hgr->hedgeNum + N_PARTITION(hgr)] = 0;
                gain += hgr->nodes[nodeListnz[i].nodeid-hgr->hedgeNum + N_WEIGHT(hgr)];
                i++;
                if (gain >= static_cast<long>(targetWeight0)) {
                    break;
                }
                if (i > sqrt(hgr->totalWeight)) {
                    break;
                }
            }
            gettimeofday(&end, NULL);
            float elapsed = (end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0);
            time += (time0 + time1 + time2) / 1000.f + elapsed;
            std::cout << "i:" << i << "\n";
            std::cout << "gain:" << gain << ", targetWeight0:" << targetWeight0 << "\n";
            std::cout << "current edge cut quality:" << computeHyperedgeCut(hgr) << "\n";
            CHECK_ERROR(hipFree(nodeListnz));
            if (gain >= static_cast<long>(targetWeight0)) {
                break;
            }
        }
        CHECK_ERROR(hipFree(count));
    }
}
