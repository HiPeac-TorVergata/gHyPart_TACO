#include "hip/hip_runtime.h"
#include <iostream>
#include "utility/utils.cuh"
#include "kernels/refinement_kernels.cuh"
#include "include/refinement_impl.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <sys/time.h>
#include <algorithm>
#include <bits/stdc++.h>


void refinement_opt3(Hypergraph* hgr, unsigned refineTo, unsigned int K, float imbalance, float& time, float& cur, int cur_iter, float ratio, int iterNum, OptionConfigs& optcfgs) {
    std::cout << __FUNCTION__ << "()===========\n";
    int nonzeroPartWeight = 0;
    // int nonzeroCnt = 0;
    // for (int i = 0; i < hgr->nodeNum; ++i) {
    //     if (hgr->nodes[i + N_PARTITION(hgr)] > 0) {
    //         nonzeroPartWeight += hgr->nodes[i + N_WEIGHT(hgr)];
    //         nonzeroCnt++;
    //     }
    // }
    // std::cout << "nonzeroCnt:" << nonzeroCnt << "\n";
    int zeroPartWeight = hgr->totalWeight - nonzeroPartWeight;
    std::cout << "totalweight:" << hgr->totalWeight << ", part0_weight:" << zeroPartWeight << ", part1_weight:" << nonzeroPartWeight << "\n";
    unsigned *zeroW, *nonzeroW;
    CHECK_ERROR(hipMallocManaged(&zeroW, sizeof(unsigned)));
    CHECK_ERROR(hipMallocManaged(&nonzeroW, sizeof(unsigned)));
    std::cout << "maxNodeWeight:" << hgr->maxWeight << ", minNodeWeight:" << hgr->minWeight << "\n";
    unsigned pass = 0;
    while (pass < refineTo) {
        int blocksize = 128;
        int gridsize = UP_DIV(hgr->hedgeNum, blocksize);
        TIMERSTART(0)
        thrust::fill(thrust::device, hgr->nodes + N_FS(hgr), hgr->nodes + N_FS(hgr) + hgr->nodeNum, 0);
        thrust::fill(thrust::device, hgr->nodes + N_TE(hgr), hgr->nodes + N_TE(hgr) + hgr->nodeNum, 0);
        initGains<<<gridsize, blocksize>>>(hgr, hgr->hedgeNum);
        TIMERSTOP(0)

        zeroW[0] = 0, nonzeroW[0] = 0;
        tmpNode *zeroNodeList, *nonzeroNodeList;
        CHECK_ERROR(hipMallocManaged(&zeroNodeList, hgr->nodeNum * sizeof(tmpNode)));
        CHECK_ERROR(hipMallocManaged(&nonzeroNodeList, hgr->nodeNum * sizeof(tmpNode)));
        gridsize = UP_DIV(hgr->nodeNum, blocksize);
        TIMERSTART(1)
        createNodeLists<<<gridsize, blocksize>>>(hgr, zeroNodeList, nonzeroNodeList, zeroW, nonzeroW);
        TIMERSTOP(1)
        std::cout << __LINE__ << ":" << zeroW[0] << ", " << nonzeroW[0] << "\n";

        thrust::device_ptr<tmpNode> zero_ptr(zeroNodeList);
        thrust::device_ptr<tmpNode> one_ptr(nonzeroNodeList);
        TIMERSTART(2)
        thrust::sort(thrust::device, zero_ptr, zero_ptr + zeroW[0], mycmp());
        thrust::sort(thrust::device, one_ptr, one_ptr + nonzeroW[0], mycmp());
        // thrust::sort(thrust::device, zeroNodeList, zeroNodeList + zeroW[0], mycmp());
        // thrust::sort(thrust::device, nonzeroNodeList, nonzeroNodeList + nonzeroW[0], mycmp());
        TIMERSTOP(2)

        unsigned workLen = zeroW[0] <= nonzeroW[0] ? 2 * zeroW[0] : 2 * nonzeroW[0];
        gridsize = UP_DIV(workLen, blocksize);
        TIMERSTART(3)
        // parallelSwapNodes<<<gridsize, blocksize>>>(hgr, zeroNodeList, nonzeroNodeList, zeroW, nonzeroW, workLen);
        TIMERSTOP(3)
        for (int i = 0; i < workLen / 2; ++i) {
            if (cur_iter > 0 && zeroNodeList[i].weight == hgr->maxWeight) { // skip heaviest node util finest level
                continue;
            }
            hgr->nodes[N_PARTITION(hgr) + zeroNodeList[i].nodeid - hgr->hedgeNum] = 1;
            hgr->nodes[N_COUNTER(hgr) + zeroNodeList[i].nodeid - hgr->hedgeNum]++;
        }
        for (int i = 0; i < workLen / 2; ++i) {
            if (cur_iter > 0 && nonzeroNodeList[i].weight == hgr->maxWeight) {
                continue;
            }
            hgr->nodes[N_PARTITION(hgr) + nonzeroNodeList[i].nodeid - hgr->hedgeNum] = 0;
            hgr->nodes[N_COUNTER(hgr) + nonzeroNodeList[i].nodeid - hgr->hedgeNum]++;
        }
        time += (time0 + time1 + time2 + time3) / 1000.f;
        cur += (time0 + time1 + time2 + time3) / 1000.f;
        std::cout << "time:" << time << " s.\n";
        pass++;
        
        CHECK_ERROR(hipFree(zeroNodeList));
        CHECK_ERROR(hipFree(nonzeroNodeList));
    }
    TIMERSTART(4)
    thrust::fill(hgr->nodes + N_COUNTER(hgr), hgr->nodes + N_COUNTER(hgr) + hgr->nodeNum, 0);
    TIMERSTOP(4)
    time += time4 / 1000.f;
    cur += time4 / 1000.f;
    std::cout << "current edge cut quality:" << computeHyperedgeCut(hgr) << "\n";
}

void rebalancing_opt3(Hypergraph* hgr, float ratio, unsigned int K, float imbalance, float& time, float& cur, int& rebalance, int cur_iter, int iterNum, OptionConfigs& optcfgs) {
    std::cout << __FUNCTION__ << "()===========\n";
    int nonzeroPartWeight = 0;
    int nonzeroCnt = 0;
    struct timeval bbeg, eend;
    gettimeofday(&bbeg, NULL);
    for (int i = 0; i < hgr->nodeNum; ++i) {
        if (hgr->nodes[i + N_PARTITION(hgr)] > 0) {
            if (cur_iter > 0 && hgr->nodes[i + N_WEIGHT(hgr)] == hgr->maxWeight) {
                continue;
            }
            nonzeroPartWeight += hgr->nodes[i + N_WEIGHT(hgr)];
            nonzeroCnt++;
        }
    }
    gettimeofday(&eend, NULL);
    float elap = (eend.tv_sec - bbeg.tv_sec) + ((eend.tv_usec - bbeg.tv_usec)/1000000.0);
    std::cout << "elapsed time: " << elap << " s.\n";
    time += elap;
    cur += elap;
    float tol    = std::max(ratio, 1 - ratio) - 1; // 10 / 45
    int total = cur_iter > 0 ? hgr->totalWeight - hgr->maxWeight : hgr->totalWeight;
    int hi = (1 + tol) * total / (2 + tol); // 55 / 100
    // const int hi = (1 + imbalance / 100) * (hgr->totalWeight / K);
    const int lo = total - hi;
    int bal      = nonzeroPartWeight;
    std::cout << "nonzeroCnt:" << nonzeroCnt << "\n";
    std::cout << bal << ", " << hi << ", " << lo << ", " << tol << ", " << ratio << "\n";
    while (1) {
        if (bal >= lo && bal <= hi) {
            break;
        }

        // int blocksize = 128;
        // int gridsize = UP_DIV(hgr->hedgeNum, blocksize);
        // TIMERSTART(0)
        // thrust::fill(thrust::device, hgr->nodes + N_FS(hgr), hgr->nodes + N_FS(hgr) + hgr->nodeNum, 0);
        // thrust::fill(thrust::device, hgr->nodes + N_TE(hgr), hgr->nodes + N_TE(hgr) + hgr->nodeNum, 0);
        // initGains<<<gridsize, blocksize>>>(hgr, hgr->hedgeNum);
        // TIMERSTOP(0)

        // unsigned *bucketcnt;
        // CHECK_ERROR(hipMallocManaged(&bucketcnt, 101 * sizeof(unsigned)));
        // unsigned *negCnt;
        // CHECK_ERROR(hipMallocManaged(&negCnt, sizeof(unsigned)));
        std::cout << "bal:" << bal << ", hi:" << hi << ", lo:" << lo << "\n";
        if (bal < lo) {
            rebalance++;
            int blocksize = 128;
            int gridsize = UP_DIV(hgr->hedgeNum, blocksize);
            TIMERSTART(0)
            thrust::fill(thrust::device, hgr->nodes + N_FS(hgr), hgr->nodes + N_FS(hgr) + hgr->nodeNum, 0);
            thrust::fill(thrust::device, hgr->nodes + N_TE(hgr), hgr->nodes + N_TE(hgr) + hgr->nodeNum, 0);
            initGains<<<gridsize, blocksize>>>(hgr, hgr->hedgeNum);
            TIMERSTOP(0)

            unsigned *bucketcnt;
            CHECK_ERROR(hipMallocManaged(&bucketcnt, 101 * sizeof(unsigned)));
            unsigned *negCnt;
            CHECK_ERROR(hipMallocManaged(&negCnt, sizeof(unsigned)));
            std::cout << "enter bal < lo branch+++++++++\n";
            // placing each node in an appropriate bucket using the gain by weight ratio
            negCnt[0] = 0;
            tmpNode *nodelistz;
            CHECK_ERROR(hipMallocManaged(&nodelistz, 101 * hgr->nodeNum * sizeof(tmpNode)));
            tmpNode *negGainlistz;
            CHECK_ERROR(hipMallocManaged(&negGainlistz, hgr->nodeNum * sizeof(tmpNode)));

            // int blocksize = 128;
            gridsize = UP_DIV(hgr->nodeNum, blocksize);
            unsigned partID = 0;
            TIMERSTART(1)
            thrust::fill(bucketcnt, bucketcnt + 101, 0);
            placeNodesInBuckets<<<gridsize, blocksize>>>(hgr, nodelistz, bucketcnt, negGainlistz, negCnt, partID);
            TIMERSTOP(1)
            
            int total_count = 0;
            unsigned min_element = INT_MAX;
            unsigned max_element = 0;
            thrust::device_ptr<tmpNode> zero_ptr(nodelistz);
            // sorting each bucket in parallel
            TIMERSTART(2)
            for (int i = 0; i < 101; ++i) {
                total_count += bucketcnt[i];
                if (bucketcnt[i] > 1) {
                    hipStream_t stream;
                    hipStreamCreateWithFlags(&stream, hipStreamDefault);
                    thrust::sort(thrust::cuda::par.on(stream), nodelistz + i * hgr->nodeNum, nodelistz + i * hgr->nodeNum + bucketcnt[i], cmpGbyW());
                    CHECK_ERROR(hipStreamSynchronize(stream));
                    hipStreamDestroy(stream);
                    if (min_element > bucketcnt[i]) min_element = bucketcnt[i];
                    if (max_element < bucketcnt[i]) max_element = bucketcnt[i];
                }
            }
            TIMERSTOP(2)
            std::cout << "totally, there are " << total_count << " waiting move candidates!!!\n";
            
            unsigned i = 0;
            unsigned j = 0;
            // now moving nodes from partition 0 to 1
            struct timeval begin, end;
            gettimeofday(&begin, NULL);
            while (j < 101) {
                if (bucketcnt[j] == 0) {
                    j++;
                    continue;
                }
                for (int k = 0; k < bucketcnt[j]; ++k) {
                    if (cur_iter > 0 && nodelistz[j * hgr->nodeNum + k].weight == hgr->maxWeight) {
                        continue;
                    }
                    hgr->nodes[nodelistz[j * hgr->nodeNum + k].nodeid-hgr->hedgeNum + N_PARTITION(hgr)] = 1;
                    bal += hgr->nodes[nodelistz[j * hgr->nodeNum + k].nodeid-hgr->hedgeNum + N_WEIGHT(hgr)];
                    if (bal >= lo) {
                        std::cout << __LINE__ << " break balance!!!\n";
                        break;
                    }
                    i++;
                    if (i > sqrt(hgr->nodeNum)) {
                        break;
                    }
                }
                if (bal >= lo) {
                    break;
                }
                if (i > sqrt(hgr->nodeNum)) {
                    break;
                }
                j++;
            }
            gettimeofday(&end, NULL);
            std::cout << "max bucket:" << max_element << "\n";
            std::cout << "min bucket:" << min_element << "\n";
            // std::sort(bucketcnt, bucketcnt + 101);
            // std::cout << "q1 bucket:" << bucketcnt[101 / 4] << ", med bucket:" << bucketcnt[101 / 2] << ", q3 bucket:" << bucketcnt[101*3/4] << "\n";
            std::cout << "Until " << j << "-th bucket, # processed moves:" << i << "\n";
            float elapsed = (end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0);
            time += (time0 + time1 + time2) / 1000.f + elapsed;
            cur += (time0 + time1 + time2) / 1000.f + elapsed;
            std::cout << "elapsed time: " << elapsed << " s.\n";
            std::cout << __LINE__ << "time:" << time << " s.\n";
            std::cout << "bal:" << bal << ", lo:" << lo << "\n";
            // std::cout << "rebalance: current edge cut quality:" << computeHyperedgeCut(hgr) << "\n";
            if (bal >= lo) {
                break;
            }
            if (i > sqrt(hgr->nodeNum)) {
                continue;
            }

            // moving nodes from nodeListzNegGain
            if (negCnt[0] == 0) {
                continue;
            }
            thrust::device_ptr<tmpNode> negzero_ptr(negGainlistz);
            TIMERSTART(3)
            thrust::sort(/*thrust::device, */negGainlistz, negGainlistz + negCnt[0], cmpGbyW());
            // thrust::sort(thrust::device, negzero_ptr, negzero_ptr + negCnt[0], cmpGbyW());
            TIMERSTOP(3)
            gettimeofday(&begin, NULL);
            for (int k = 0; k < negCnt[0]; ++k) {
                hgr->nodes[negGainlistz[k].nodeid-hgr->hedgeNum + N_PARTITION(hgr)] = 1;
                bal += hgr->nodes[negGainlistz[k].nodeid-hgr->hedgeNum + N_WEIGHT(hgr)];
                if (bal >= lo) {
                    std::cout << __LINE__ << " break balance!!!\n";
                    break;
                }
                i++;
                if (i > sqrt(hgr->nodeNum)) {
                    break;
                }
            }
            gettimeofday(&end, NULL);
            elapsed = (end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0);
            time += time3 / 1000.f + elapsed;
            cur += time3 / 1000.f + elapsed;
            std::cout << "@elapsed time: " << elapsed << " s.\n";
            std::cout << __LINE__ << "@time:" << time << " s.\n";
            std::cout << "@bal:" << bal << ", lo:" << lo << "\n";
            // std::cout << "@rebalance: current edge cut quality:" << computeHyperedgeCut(hgr) << "\n";
            CHECK_ERROR(hipFree(nodelistz));
            CHECK_ERROR(hipFree(negGainlistz));
            CHECK_ERROR(hipFree(bucketcnt));
            if (bal >= lo) {
                break;
            }
        } else { // bal > hi
            rebalance++;
            int blocksize = 128;
            int gridsize = UP_DIV(hgr->hedgeNum, blocksize);
            TIMERSTART(0)
            thrust::fill(thrust::device, hgr->nodes + N_FS(hgr), hgr->nodes + N_FS(hgr) + hgr->nodeNum, 0);
            thrust::fill(thrust::device, hgr->nodes + N_TE(hgr), hgr->nodes + N_TE(hgr) + hgr->nodeNum, 0);
            initGains<<<gridsize, blocksize>>>(hgr, hgr->hedgeNum);
            TIMERSTOP(0)

            unsigned *bucketcnt;
            CHECK_ERROR(hipMallocManaged(&bucketcnt, 101 * sizeof(unsigned)));
            unsigned *negCnt;
            CHECK_ERROR(hipMallocManaged(&negCnt, sizeof(unsigned)));
            std::cout << "enter bal > hi branch+++++++++\n";
            // placing each node in an appropriate bucket using the gain by weight ratio
            negCnt[0] = 0;
            tmpNode *nodelisto;
            CHECK_ERROR(hipMallocManaged(&nodelisto, 101 * hgr->nodeNum * sizeof(tmpNode)));
            tmpNode *negGainlisto;
            CHECK_ERROR(hipMallocManaged(&negGainlisto, hgr->nodeNum * sizeof(tmpNode)));
            
            // int blocksize = 128;
            gridsize = UP_DIV(hgr->nodeNum, blocksize);
            unsigned partID = 1;
            TIMERSTART(1)
            thrust::fill(bucketcnt, bucketcnt + 101, 0);
            placeNodesInBuckets<<<gridsize, blocksize>>>(hgr, nodelisto, bucketcnt, negGainlisto, negCnt, partID);
            TIMERSTOP(1)
            
            // unsigned min_element = INT_MAX;
            // unsigned max_element = 0;
            thrust::device_ptr<tmpNode> one_ptr(nodelisto);
            // sorting each bucket in parallel
            TIMERSTART(2)
            for (int i = 0; i < 101; ++i) {
                if (bucketcnt[i] > 1) {
                    hipStream_t stream;
                    hipStreamCreateWithFlags(&stream, hipStreamDefault);
                    thrust::sort(thrust::cuda::par.on(stream), nodelisto + i * hgr->nodeNum, nodelisto + i * hgr->nodeNum + bucketcnt[i], cmpGbyW());
                    CHECK_ERROR(hipStreamSynchronize(stream));
                    hipStreamDestroy(stream);
                    // if (min_element > bucketcnt[i]) min_element = bucketcnt[i];
                    // if (max_element < bucketcnt[i]) max_element = bucketcnt[i];
                }
            }
            TIMERSTOP(2)

            unsigned i = 0;
            unsigned j = 0;
            // now moving nodes from partition 0 to 1
            struct timeval begin, end;
            gettimeofday(&begin, NULL);
            while (j < 101) {
                if (bucketcnt[j] == 0) {
                    j++;
                    continue;
                }
                for (int k = 0; k < bucketcnt[j]; ++k) {
                    if (cur_iter > 0 && nodelisto[j * hgr->nodeNum + k].weight == hgr->maxWeight) {
                        continue;
                    }
                    hgr->nodes[nodelisto[j * hgr->nodeNum + k].nodeid-hgr->hedgeNum + N_PARTITION(hgr)] = 0;
                    bal -= hgr->nodes[nodelisto[j * hgr->nodeNum + k].nodeid-hgr->hedgeNum + N_WEIGHT(hgr)];
                    if (bal <= hi) {
                        break;
                    }
                    i++;
                    if (i > sqrt(hgr->nodeNum)) {
                        break;
                    }
                }
                if (bal <= hi) {
                    break;
                }
                if (i > sqrt(hgr->nodeNum)) {
                    break;
                }
                j++;
            }
            gettimeofday(&end, NULL);
            // std::cout << "max bucket:" << max_element << "\n";
            // std::cout << "min bucket:" << min_element << "\n";
            // std::sort(bucketcnt, bucketcnt + 101);
            // std::cout << "q1 bucket:" << bucketcnt[101 / 4] << ", med bucket:" << bucketcnt[101 / 2] << ", q3 bucket:" << bucketcnt[101*3/4] << "\n";
            std::cout << "Until " << j << "-th bucket, # processed moves:" << i << "\n";
            float elapsed = (end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0);
            time += (time0 + time1 + time2) / 1000.f + elapsed;
            cur += (time0 + time1 + time2) / 1000.f + elapsed;
            std::cout << "elapsed time: " << elapsed << " s.\n";
            std::cout << __LINE__ << "time:" << time << " s.\n";
            std::cout << "bal:" << bal << ", lo:" << lo << "\n";
            // std::cout << "rebalance: current edge cut quality:" << computeHyperedgeCut(hgr) << "\n";
            if (bal <= hi) {
                break;
            }
            if (i > sqrt(hgr->nodeNum)) {
                continue;
            }

            // moving nodes from nodeListzNegGain
            if (negCnt[0] == 0) {
                continue;
            }
            thrust::device_ptr<tmpNode> negone_ptr(negGainlisto);
            TIMERSTART(3)
            thrust::sort(/*thrust::device, */negGainlisto, negGainlisto + negCnt[0], cmpGbyW());
            // thrust::sort(thrust::device, negone_ptr, negone_ptr + negCnt[0], cmpGbyW());
            TIMERSTOP(3)
            gettimeofday(&begin, NULL);
            for (int k = 0; k < negCnt[0]; ++k) {
                hgr->nodes[negGainlisto[k].nodeid-hgr->hedgeNum + N_PARTITION(hgr)] = 0;
                bal -= hgr->nodes[negGainlisto[k].nodeid-hgr->hedgeNum + N_WEIGHT(hgr)];
                if (bal <= hi) {
                    break;
                }
                i++;
                if (i > sqrt(hgr->nodeNum)) {
                    break;
                }
            }
            gettimeofday(&end, NULL);
            elapsed = (end.tv_sec - begin.tv_sec) + ((end.tv_usec - begin.tv_usec)/1000000.0);
            time += time3 / 1000.f + elapsed;
            cur += time3 / 1000.f + elapsed;
            std::cout << "@elapsed time: " << elapsed << " s.\n";
            std::cout << __LINE__ << "@time:" << time << " s.\n";
            std::cout << "@bal:" << bal << ", lo:" << lo << "\n";
            // std::cout << "@rebalance: current edge cut quality:" << computeHyperedgeCut(hgr) << "\n";
            CHECK_ERROR(hipFree(nodelisto));
            CHECK_ERROR(hipFree(negGainlisto));
            CHECK_ERROR(hipFree(bucketcnt));
            if (bal <= hi) {
                break;
            }
        }
        // CHECK_ERROR(hipFree(bucketcnt));
    }
    std::cout << "rebalance: current edge cut quality:" << computeHyperedgeCut(hgr) << "\n";
}

